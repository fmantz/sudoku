
#include <hip/hip_runtime.h>
#include <iostream>
#include <stdlib.h>
#include <stdio.h>

__global__ void cuda_hello(){
    printf("Hello World from GPU! %d\n", threadIdx.x*gridDim.x);
}

//simple dummy hello world:
extern "C"  //prevent C++ name mangling!
int solve_on_cuda(char p[3]){ //library method

    for(int i = 0; i < 3; i++) {
        printf("parameter %d \n", p[i]);
    }

    printf("Hello World from CPU!\n");
//    cuda_hello<<<500,1024>>>();
    cuda_hello<<<1,10>>>();
    hipDeviceSynchronize();
    return 0;
}
