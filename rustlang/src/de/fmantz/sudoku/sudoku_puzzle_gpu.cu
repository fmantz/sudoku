
#include <hip/hip_runtime.h>
#include <iostream>
#include <stdlib.h>
#include <stdio.h>

__global__ void cuda_hello(){
    printf("Hello World from GPU! %d\n", threadIdx.x*gridDim.x);
}

struct SudokuPuzzleData {
    bool my_is_solvable;
    bool my_is_solved;
    char puzzle[81];
};

//simple dummy hello world:
extern "C"  //prevent C++ name mangling!
int solve_on_cuda(SudokuPuzzleData p[], int count){ //library method

    for(int i = 0; i < count; i++) {
        SudokuPuzzleData current = p[i];
        for(int j = 0; j < 81; j++) {
            if(j % 9 == 0){
              printf("\n");
            }
            printf("%d", current.puzzle[j]);
        }
        printf("\n-----------");
    }

    printf("Hello World from CPU!\n");
//    cuda_hello<<<500,1024>>>();
    cuda_hello<<<1,10>>>();
    hipDeviceSynchronize();
    return 0;
}
